#include "cumatrix.h"
#include <stdio.h>


void timeitCU(double *matrix, size_t n, void (*testFunc) (double *, size_t)) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    testFunc(matrix, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("CUDA time: %.3f ms\n\n", milliseconds);
}

void testGaussDetCU(double *matrix, size_t n) {
    double det = determinantGauss(matrix, n);
    printf("Det (Gauss): %llf\n", det);
}

void testGaussInvCU(double *matrix, size_t n) {
    double *revM = inversedFromGauss(matrix, n);
    if (!revM) {
        printf("det = 0 or idk\n");
        hipFree(matrix);
        return;
    }
    printf("Inverted matrix (Gauss):\n");
    //printMatrix(revM, n);
    
    checkCuda( hipFree(revM) );
}

void testLUDetCU(double *matrix, size_t n) {
    double *l = allocateMatrix(n);
    double *u = allocateMatrix(n);

    initL<<<getGridDim(n), THREADS_PER_BLOCK>>>(l, n);
    hipDeviceSynchronize();

    LUdecomposition(matrix, l, u, n);

    double det = determinantLU(u, n);
    printf("Det (LU): %llf\n", det);
    
    checkCuda( hipFree(l) ); 
    checkCuda( hipFree(u) ); 
}

void testLUInvCU(double *matrix, size_t n) {
    double *l = allocateMatrix(n);
    double *u = allocateMatrix(n);

    initL<<<getGridDim(n), THREADS_PER_BLOCK>>>(l, n);
    hipDeviceSynchronize();

    LUdecomposition(matrix, l, u, n);

    double *inversed = allocateMatrix(n);
    double *yAll = allocateMatrix(n);
    inversedFromLU<<<getGridDim(n), THREADS_PER_BLOCK>>>(l, u, inversed, yAll, n);
    hipDeviceSynchronize();

    printf("Inverted matrix (LU):\n");
    //printMatrix(inverted, n);
    
    checkCuda( hipFree(l) );
    checkCuda( hipFree(u) );
    checkCuda( hipFree(yAll) );
    checkCuda( hipFree(inversed) );
}

int main() {
    size_t n;
    double *matrix;
    inputMatrix(&matrix, &n);

    //
    //timeitCU(matrix, n, testGaussDetCU); 
    //timeitCU(matrix, n, testGaussInvCU); 
    timeitCU(matrix, n, testLUDetCU); 
    timeitCU(matrix, n, testLUInvCU); 
    //

    hipFree(matrix); 
    return 0;
}